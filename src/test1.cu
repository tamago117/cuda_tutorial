#include "hip/hip_runtime.h"
//reference http://tecsingularity.com/cuda/cudaprogram1/

#include <stdio.h>
#include <iostream>
#include <sstream>
#include <time.h>
#include <hip/hip_runtime.h>

bool to_bool(const char* str)
{
    std::istringstream is( str );
    bool result;
    is >> std::boolalpha >> result;
    return result;
}


// GPUで計算する際の関数
__global__ void gpu_function(float *d_x, float *d_y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	d_y[i] = sin(d_x[i]) * sin(d_x[i]) + cos(d_x[i]) * cos(d_x[i]);
}

// CPUで計算する際の関数
void cpu_function(int n, float *x, float *y)
{
	for (int i = 0; i < n; i++) {
		y[i] = sin(x[i]) * sin(x[i]) + cos(x[i]) * cos(x[i]);
	}
}

// main function
int main(int argc, char** argv)
{
	if(argc < 2){
		std::cout << "Usage: ./test1 [true/false]" << std::endl;
		return 0;
	}

	bool GPU = to_bool(argv[1]);
	if(GPU){
		printf("GPU mode\n");
	}else{
		printf("CPU mode\n");
	}

	int N = 100000000;
	float *host_x, *host_y, *dev_x, *dev_y;
	
	// CPU側の領域確保
	host_x = (float*)malloc(N * sizeof(float));
	host_y = (float*)malloc(N * sizeof(float));

	// 乱数値を入力する
	for (int i = 0; i < N; i++) {
		host_x[i] = rand();
	}

	int start = clock();

	if (GPU == true) {

		// デバイス(GPU)側の領域確保
		hipMalloc(&dev_x, N * sizeof(float));
		hipMalloc(&dev_y, N * sizeof(float));

		// CPU⇒GPUのデータコピー
		hipMemcpy(dev_x, host_x, N * sizeof(float), hipMemcpyHostToDevice);

		// GPUで計算
		gpu_function << <(N + 255) / 256, 256 >> >(dev_x, dev_y);

		// GPU⇒CPUのデータコピー
		hipMemcpy(host_y, dev_y, N * sizeof(float), hipMemcpyDeviceToHost);

	}
	else {
		// CPUで計算
		cpu_function(N, host_x, host_y);
	}

	int end = clock();

	// 計算が正しく行われているか確認
	float sum = 0.0f;
	for (int j = 0; j < N; j++) {
		sum += host_y[j];
	}
	std::cout << sum << std::endl;

	// 最後に計算時間を表示
	std::cout << "clock : "<< (end - start) << std::endl;

	return 0;
}